#include "hip/hip_runtime.h"
#include "KDTree.cuh"
#include <climits>
#include <cfloat>
#include <algorithm>
#include <iostream>
#include <cmath>
#include <utility>
#include <numeric>
#include <thrust/host_vector.h>

namespace
{
#if KDTREE_TRACE

    template<typename... T>
    void trace(T&& ... args)
    {
        ((std::cout << args << " "), ...) << std::endl;
    }

    template<typename T>
    void print_vector(const std::string& title, const thrust::device_vector<T>& t)
    {
        thrust::host_vector<T> v(t);

        std::cout << title << ": ";
        for(int i = 0; i < v.size(); i++)
        {
            std::cout << v[i] << ", ";
        }
        std::cout << std::endl;
    }

#else
    template<typename... T> void trace(T&&...) {}
    template<typename... T> void print_vector(T&&...) {}
#endif
}

AABB KDTreeBase::computeBoundingBox(const Mesh& mesh)
{
    // Computing the AABB of a mesh is just the AABB of all points because an AABB is convex

    AABB res{};

    if(!mesh.empty())
    {
        const auto& firstPoint = mesh[0].points[0];

        // Initialize the bounding box to a point
        for(int dim = 0; dim < 3; dim++)
        {
            res.min[dim] = firstPoint[dim];
            res.max[dim] = firstPoint[dim];
        }

        // Grow the bounding box for each point if needed
        for(const auto& triangle: mesh)
        {
            for(const auto& point: triangle.points)
            {
                for(int dim = 0; dim < 3; dim++)
                {
                    store_min(res.min[dim], point[dim]);
                    store_max(res.max[dim], point[dim]);
                }
            }
        }
    }

    return res;
}

KDTree::KDTree(const Mesh& mesh, Heuristics heuristics)
    : m_mesh(mesh),
      m_rootAABB(computeBoundingBox(mesh)),
      m_heuristics(heuristics)
{
    init();
}

namespace
{
    /**
     * For the original version on the CPU,
     * We can't use recursive functions, because the OS stack is too small for our needs in recursion.
     * Its also more optimized to not use recursive functions (no context argument needed).
     *
     * Also for the GPU version, we offload each task on the GPU.
     * Treat the members as the parameters of the recursive function.
     * Also, we iterate BFS and not DFS.
     */
    struct TaskData
    {
        int dim; ///< The dimension of the split
        KDTree::NodeID nodeID; ///< The node, to fill
        AABB aabb; ///< The bounding box of the node.
        int level; ///< Current level of depth. Used for termination condition.
        int inputMeshOff; ///< Remaining candidates for this node, pointing to TaskOp::inputMesh
        int inputMeshSize; ///< Count of input mesh for this node
        int outputMeshOff; ///< Preallocated output for this mesh split, pointing to TaskOp::outputMesh,, of size x2 inputMeshSize, first half is near node, second half is far node
    };

    /**
     * Thrust operation to fill the current level.
     * All member pointers point to the device memory.
     */
    struct TaskOp : KDTreeBase
    {
        /**
         * Pointer to all the output tasks.
         * Output array filled to store the new/outputs/children tasks.
         * It has always enough place (x2 size of input tasks).
         * It should be synchronized to avoid data race.
         * You can just fetch-and-add (atomicAdd in CUDA) the current size to reserve some offset in the array.
         * The size is just a single integer, as a pointer to save it on the host side.
         */
        TaskData *outputTasks;
        int *outputTasksSize;

        /**
         * Front of the KDTree::m_leavesBuffer array.
         * Like outputTasks, filling this vector should be synchronized with fetch-and-add.
         * The size is just a single integer, as a pointer to save it on the host side.
         */
        Triangle::ID *leavesBuffer;
        int *leavesBufferSize;

        /**
         * Front of the KDTree::m_node array.
         * No need for synchronization, each kernel operates on a different node.
         */
        Node *nodes;

        /**
         * Same as the local variable in KDTree::init().
         * Temporary buffers during the tree construction.
         */
        const Triangle::ID *inputTriangles;
        Triangle::ID *outputTriangles;

        /**
         * Same as the local variable in KDTree::init().
         * Should be synchronized.
         */
        int *totalOutputTriangles;

        /**
         * Same as KDTree::m_mesh
         */
        const Triangle *mesh;

        Heuristics heuristics;

        /**
         * @brief Split a set of triangles in near and far sets.
         *
         * @param subMesh The mesh to split.
         * @param subMeshSize The count of triangles in the mesh.
         * @param axe The position of the split straight line.
         * @param outputsOrig
         *      Where to store the split mesh.
         *      outputs[NEAR] will store the near mesh.
         *      output[FAR] will store the far mesh.
         *      Some triangles can be stored on both sides.
         *      The array must be preallocated and have enough place.
         *      That means, each output should have at least the size of the input mesh because
         *      we don't know and maybe all triangles will belong to both children.
         *
         * @param outputSizes
         *      outputSizes[NEAR] will store the size of the near output mesh.
         *      outputSizes[NEAR] will store the size of the far output mesh.
         */
        CUDA_BOTH void
        split(const Triangle::ID *subMesh, int subMeshSize, const Line& axe, Triangle::ID *const outputsOrig[2],
              int outputSizes[2]);

        __device__ void operator()(const TaskData& taskData);
    };
}

void KDTree::init()
{
    const size_t bytesBuffer = 10 * MB;

    // Allocate the maximum possible size for the leaves node
    // If it is outreached then undefined behaviour; pointers may be invalided on reallocation
    // And leaves mesh will point to undefined value
    m_leavesBuffer.resize(bytesBuffer / sizeof(m_leavesBuffer[0]));


    // Allocate the maximum number of node
    // Also initialize to zero (empty node)
    m_nodes.resize(getMaxNodesCount());

    // We use a vector just because we need the clear() method which is more effecient
    // than a pop() loop, or '= {}' because it may reallocate memory
    thrust::device_vector<TaskData> inputTasks;

    // Split recursively in x, y, z, x, y, z...
    // Split at the center

    // dim axis -->

    // 0 --------- aabb[dim].min --------------------------- aabb[dim].max --------- +inf
    // ------------------|--------------------|-------------------|-------------------
    // ---------------------- left node ----------- right node -----------------------

    //                    <----------------->    <--------------->
    // splitDistance:        if left                 if right

    {
        // Generator (master) thread current level
        int generatorLevel = 0;

        // The temporary working buffer to store the triangles for the current level (input),
        // And the next level (output)
        // We don't know a reasonable upper bound of the size of the vectors, or a very high upper bounds impracticable
        // to reserve: the level L has at maximum 2^L * mesh.size() triangles, which is very high for deep levels.
        thrust::device_vector<Triangle::ID> inputTriangles;
        thrust::device_vector<Triangle::ID> outputTriangles;

        // To split the task children, allocate memory at each level only once globally for the triangles
        // Each task has a unique array associated to it starting at task.inputMesh pointing in same
        // memory inside inputTriangles.
        // There is as well an output buffer mesh.outputMesh pointing inside an offset in outputTriangles.
        // This temporary working "buffer" for the task is of size 2x the count of triangles of the task
        // That means the working buffer will always have enough space to store the children triangles even
        // if both have all the mesh (as a note, this case is very inefficient). We could optimize it and stop
        // Recursion for example if more than X% of triangles are on both side.
        // The working buffer is split in two contiguous array to store the children near and far triangles.
        // There is no race condition on the input variable because each task may only access its own part of the array.
        // There is also no race condition on the output because each task has its own part to the output array.

        // At first, there is only one task (the root)
        // And all the triangles are contained once in the root node
        inputTriangles.resize(m_mesh.size());
        thrust::sequence(inputTriangles.begin(), inputTriangles.end(), 0); // Initial candidates is the entire mesh

        // At the very most both near and far children of the root store all the mesh
        outputTriangles.resize(inputTriangles.size() * 2);

        {
            // First task
            inputTasks.push_back(TaskData{
                .dim = 0,
                .nodeID = 0,
                .aabb = m_rootAABB,
                .level = 0,
                .inputMeshOff = 0,
                .inputMeshSize = static_cast<int>(inputTriangles.size()),
                .outputMeshOff = 0
            });
        }

        thrust::device_vector<TaskData> outputTasks;

        // Array of one element as a managed data on the device
        thrust::device_vector<int> leavesBufferSize(1, 0);

        while(!inputTasks.empty())
        {
            // Run all the stack tasks at once, then wait all
            trace("Spawn", inputTasks.size(), "tasks for level", generatorLevel);
            print_vector("inputTriangles", inputTriangles);

            // Run all tasks
            // First get all tasks to avoid a race condition on the stack,
            // because a task may push() to the stack, possibly immediately
            // At maximum each task will generate 2 new tasks (if non-leaf node)
            outputTasks.resize(inputTasks.size() * 2);
            thrust::device_vector<int> outputTasksSize(1, 0);

            // Total count of triangles outputs for the current level, which is also
            // the total count of input triangles for the next level.
            // Filled as things progress by the tasks (atomically to avoid race condition)
            // Also permit to know where the offset should be for each task in the output.
            // Just use thrust::device_vector of size 1 like a std::unique_ptr on the device to manage memory
            thrust::device_vector<int> totalOutputTriangles(1, 0);

            TaskOp op;
            op.outputTasks = thrust::raw_pointer_cast(outputTasks.data());
            op.outputTasksSize = thrust::raw_pointer_cast(outputTasksSize.data());
            op.leavesBuffer = thrust::raw_pointer_cast(m_leavesBuffer.data());
            op.leavesBufferSize = thrust::raw_pointer_cast(leavesBufferSize.data());
            op.nodes = thrust::raw_pointer_cast(m_nodes.data());
            op.inputTriangles = thrust::raw_pointer_cast(inputTriangles.data());
            op.outputTriangles = thrust::raw_pointer_cast(outputTriangles.data());
            op.totalOutputTriangles = thrust::raw_pointer_cast(totalOutputTriangles.data());
            op.mesh = thrust::raw_pointer_cast(m_mesh.data());
            op.heuristics = m_heuristics;

            thrust::for_each(inputTasks.begin(), inputTasks.end(), op);

            trace("Output buffer size: ", outputTriangles.size() * sizeof(outputTriangles[0]));

            // Double buffering of temporary split buffer
            using std::swap;
            swap(inputTriangles, outputTriangles);

            outputTasks.resize(outputTasksSize[0]);
            swap(inputTasks, outputTasks);

            // Allocate the next output buffer
            // We don't care of the content as it will be overwritten,
            // if there is enough space no reallocation will occur wich is good
            // The size of the next output buffer is upper bounded by twice the count of next total inputs.
            outputTriangles.resize(totalOutputTriangles[0] * 2);

            generatorLevel++;
        }

        m_leavesBuffer.resize(leavesBufferSize[0]);
    }
}

CUDA_BOTH
void TaskOp::split(const Triangle::ID *subMesh, int subMeshSize, const Line& axe, Triangle::ID *const outputsOrig[2],
                   int outputSizes[2])
{
    // Save locally to not modify original pointer
    Triangle::ID *outputs[2];
    outputs[NEAR] = outputsOrig[NEAR];
    outputs[FAR] = outputsOrig[FAR];


    // Iterate all triangles,
    // We can't split them as they can belong to both sides
    for(int i = 0; i < subMeshSize; i++)
    {
        const auto& triangleID = subMesh[i];
        const Triangle& triangle = mesh[triangleID];

        // If all points of the triangle are on one side, the triangle is not colliding with the other side (because
        // triangle and AABB are convex shapes).

        const auto side = axe.query(triangle.points[0]);
        if(side == axe.query(triangle.points[1]) && side == axe.query(triangle.points[2]))
        {
            // All points are on the same side
            // So the triangle is on one side
            *(outputs[side]++) = (triangleID);
        }
        else
        {
            // All points are not on the same side
            // So the triangle is one both side
            *(outputs[NEAR]++) = (triangleID);
            *(outputs[FAR]++) = (triangleID);
        }
    }

    for(int i = 0; i < 2; i++)
    {
        outputSizes[i] = static_cast<int>((outputs[i] - outputsOrig[i]));
    }
}

__device__
void TaskOp::operator()(const TaskData& taskData)
{
    const NodeID nodeID = taskData.nodeID;
    Node& node = nodes[nodeID];
    const int dim = taskData.dim;
    const AABB aabb = taskData.aabb;
    const int level = taskData.level;

    // Stop condition
    // FOR TRIANGLES: it's not guaranteed we can have less a given number of triangles, so we always should
    // Stop on a max. level
    if(taskData.inputMeshSize > heuristics.maxNodeSize && level < heuristics.maxLevel)
    {
        // Node must be split, split here

        node.header.dim = dim;
        node.header.hasChildren = true;

        // We split at the center of the parent AABB
        node.p = (aabb.max[dim] + aabb.min[dim]) / 2.0f;

        AABB aabbs[2];
        aabbs[NEAR] = aabb;
        aabbs[NEAR].max[dim] = node.p;
        aabbs[FAR] = aabb;
        aabbs[FAR].min[dim] = aabbs[NEAR].max[dim];

        int outputsOff[2];
        outputsOff[NEAR] = taskData.outputMeshOff;
        outputsOff[FAR] = taskData.outputMeshOff + taskData.inputMeshSize;

        Triangle::ID *outputs[2];
        outputs[NEAR] = &outputTriangles[outputsOff[NEAR]];
        outputs[FAR] = &outputTriangles[outputsOff[FAR]];

        int outputSizes[2];

        // COSTLY SPLIT in preallocated memory
        split(&inputTriangles[taskData.inputMeshOff], taskData.inputMeshSize, node.line(), outputs,
              outputSizes);

        int nextDim;
        switch(heuristics.dim)
        {
            case Heuristics::DIM_2D:
                nextDim = (dim + 1) % 2;
                break;

            case Heuristics::DIM_3D:
                nextDim = (dim + 1) % 3;
                break;
        }

        NodeID childrenIDs[2];
        childrenIDs[NEAR] = 2 * nodeID + 1; // "Left child" (near)
        childrenIDs[FAR] = 2 * nodeID + 2; // "Right child" (far)

        // Reserve two new tasks indices in the list
        const int childTaskOffset = atomicAdd(outputTasksSize, 2);

        // Add the sum of new ids to the total
        // Also get the current available offset of triangles
        const int currentOutputTriangles = atomicAdd(totalOutputTriangles, outputSizes[NEAR] + outputSizes[FAR]);

        // Compute the children output offsets
        // We multiply by 2 because the global next level output buffer will be twice as large as next level input buffer
        int childOutputMeshOffs[2];
        childOutputMeshOffs[NEAR] = currentOutputTriangles * 2;
        childOutputMeshOffs[FAR] = (currentOutputTriangles * 2) + (outputSizes[NEAR] * 2);

        for(int s = 0; s < 2; s++) // for NEAR and FAR
        {
            const int ti = (childTaskOffset + s);
            outputTasks[ti] = TaskData{
                .dim = nextDim,
                .nodeID = childrenIDs[s],
                .aabb = aabbs[s],
                .level = level + 1,
                .inputMeshOff = outputsOff[s],
                .inputMeshSize = outputSizes[s],
                .outputMeshOff = childOutputMeshOffs[s],
            };
        }
    }
    else
    {
        // Leaf node
        // Store the final mesh in the leaf node

        // Reserve a sub-array in the buffer
        const int off = atomicAdd(leavesBufferSize, taskData.inputMeshSize);

        // Copy all the inputs to the buffer
        for(int i = 0; i < taskData.inputMeshSize; i++)
        {
            leavesBuffer[off + i] = inputTriangles[taskData.inputMeshOff + i];
        }

        node.mesh = &leavesBuffer[off];
        node.meshSize = taskData.inputMeshSize;

        // DO NOT increment totalOutputTriangles
        // Because this variable is used to compute the next output size,
        // but as this is a leaf there is no child node so no need for output for this node for the next level.
    }
}

namespace
{
    CUDA_BOTH float my_saturate(float x)
    {
#ifdef __CUDA_ARCH__
        return __saturatef(x);
#else
        return std::clamp(x, 0.0f, 1.0f);
#endif
    }
}

CUDA_BOTH Point
KDTreeBase::findClosestPointOnTriangle(const Point& query, const Triangle& triangle)
{
    // https://stackoverflow.com/a/32255438/5110937

    auto edge0 = triangle.points[1] - triangle.points[0];
    auto edge1 = triangle.points[2] - triangle.points[0];
    auto v0 = triangle.points[0] - query;

    float a = dot(edge0, edge0);
    float b = dot(edge0, edge1);
    float c = dot(edge1, edge1);
    float d = dot(edge0, v0);
    float e = dot(edge1, v0);

    float det = a * c - b * b;
    float s = b * e - c * d;
    float t = b * d - a * e;

    if(s + t < det)
    {
        if(s < 0.f)
        {
            if(t < 0.f)
            {
                if(d < 0.f)
                {
                    s = my_saturate(-d / a);
                    t = 0.f;
                }
                else
                {
                    s = 0.f;
                    t = my_saturate(-e / c);
                }
            }
            else
            {
                s = 0.f;
                t = my_saturate(-e / c);
            }
        }
        else if(t < 0.f)
        {
            s = my_saturate(-d / a);
            t = 0.f;
        }
        else
        {
            float invDet = 1.f / det;
            s *= invDet;
            t *= invDet;
        }
    }
    else
    {
        if(s < 0.f)
        {
            float tmp0 = b + d;
            float tmp1 = c + e;
            if(tmp1 > tmp0)
            {
                float numer = tmp1 - tmp0;
                float denom = a - 2 * b + c;
                s = my_saturate(numer / denom);
                t = 1 - s;
            }
            else
            {
                t = my_saturate(-e / c);
                s = 0.f;
            }
        }
        else if(t < 0.f)
        {
            if(a + d > b + e)
            {
                float numer = c + e - b - d;
                float denom = a - 2 * b + c;
                s = my_saturate(numer / denom);
                t = 1 - s;
            }
            else
            {
                s = my_saturate(-e / c);
                t = 0.f;
            }
        }
        else
        {
            float numer = c + e - b - d;
            float denom = a - 2 * b + c;
            s = my_saturate(numer / denom);
            t = 1.f - s;
        }
    }

    return triangle.points[0] + edge0 * s + edge1 * t;
}