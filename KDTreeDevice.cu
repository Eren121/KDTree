#include "hip/hip_runtime.h"
#include "KDTreeDevicePtr.cuh"

KDTreeDevicePtr::KDTreeDevicePtr(const KDTree& tree)
    : m_rootAABB(tree.m_rootAABB),
      m_mesh(tree.m_mesh),
      m_nodes(tree.m_nodes),
      m_leavesBuffer(tree.m_leavesBuffer),
      m_maxLevel(tree.m_heuristics.maxLevel),
      m_totalLeafNodes(tree.m_totalLeafNodes)
{
}

__device__
NPQueryRet KDTreeDevicePtr::findNearestPointOnMesh(const Point& pos) const
{
    NPQueryRet ret{};

    float currentDist = FLT_MAX;

    searchRecursive(pos, currentDist, ret.id, ret.point);

    return ret;
}

__device__
void KDTreeDevicePtr::searchRecursive(const Point& pos,
                                      float& currentDist,
                                      Triangle::ID& currentID,
                                      Point& currentPoint) const
{
    // This should be defined as the maximum level of the array
    constexpr int MAX_STACK_SIZE = 32;
    assert(MAX_STACK_SIZE > m_maxLevel);

    // Recursive may not work well with CUDA
    // We have to do it ourselves
    // Also it is better optimized as we don't pass the references variables because we already have access to them
    struct StackEntry
    {
        NodeID nodeID;
    };
    StackEntry stack[MAX_STACK_SIZE];

    // Pointer to the current entry in the stack
    // We know it is empty when the pointers are equals
    StackEntry *top = stack;

    // Initialize the root stack entry
    // And push to the stack
    // entry is like the end iterator of the stack
    top->nodeID = 0;
    top++;

    // While the stack is not empty
    while(top != stack)
    {
        // Pop from the stack
        top--;

        const NodeID nodeID = top->nodeID;
        const Node& node = m_nodes[nodeID];

        // Are we on a leaf?
        if(node.leaf())
        {
            // We are on a leaf
            // Search brute force into the leaf node
            for(int i = 0; i < node.meshSize; i++)
            {
                const auto& triangleID = node.mesh[i];
                const auto nearestPtOnTriangle = findClosestPointOnTriangle(pos, m_mesh[triangleID]);
                const float d = nearestPtOnTriangle.distanceSquared(pos);
                if(d < currentDist)
                {
                    currentDist = d;
                    currentID = triangleID;
                    currentPoint = nearestPtOnTriangle;
                }
            }
        }
        else
        {
            NodeID front, back;
            const Line& split = node.line();

            NodeID childrenIDs[2];
            childrenIDs[NEAR] = 2 * nodeID + 1; // "Left child" (near)
            childrenIDs[FAR] = 2 * nodeID + 2; // "Right child" (far)

            // Which side I am?
            switch(split.query(pos))
            {
                case NEAR:
                    // Pos is on the near side
                    front = childrenIDs[NEAR];
                    back = childrenIDs[FAR];
                    break;

                case FAR:
                    // Pos is on the far side
                    front = childrenIDs[FAR];
                    back = childrenIDs[NEAR];
                    break;
            }

            // Push front entry
            top->nodeID = front;
            top++;
            assert((top - stack) <= MAX_STACK_SIZE);

            // If the current closest point is closer than the closest point of the back face, no need to search in the back
            // face because it will be always further.
            // If so, we save half of the time for the current node
            const float backDist = fabsf(split.p - pos[split.dim]);
            // Do not forget currentDist is squared
            if(backDist * backDist <= currentDist)
            {
                // If it can be closer, search also in this node

                // Push back entry
                top->nodeID = back;
                top++;
                assert((top - stack) <= MAX_STACK_SIZE);
            }
        }
    }
}
